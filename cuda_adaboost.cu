//ly2352, Lu Yang, Adaboost, Host version
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <hip/hip_runtime.h>
int nums = 200,cols = 256;
float **usps;
float *w;
int *y;

struct pars{
    int return_j;
    float theta;
    int return_m;
};

__global__ void
vectorAdd(const float *A, const float *B, float *C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    //printf("blockDim.x is %d,blockIdx.x is %d, threadIdx.x %d \n", blockDim.x , blockIdx.x , threadIdx.x);
    if (i < numElements)
    {
        C[0] += A[i] + B[i];
    }
}

void train(struct pars* pars_p){
  int cur_j = 0,cur_theta = 0,cur_m = 0;
  float cur_min = 100000.0;
  for (int j = 0;j<cols;j++){
    float *vec = usps[j];
    float minimal = 100000.0;
    int cur_i = 0,sel_m= 0;
    for(int i=0;i<nums;i++){
      float boundary = vec[i];
      float err = 0.0,err1 = 0.0,err2 = 0.0,sum_w = 0;
      int m = 0;
      for(int z=0;z<nums;z++){
        err1 += w[z] * ((vec[z]<=boundary) ^ (y[z]==-1));
        err2 += w[z] * ((vec[z]<=boundary) ^ (y[z]==1));
        sum_w += w[z];
      }
      
      if(err1<err2){
        err = err1/sum_w;
        m = 1;
      }else{
        err = err2/sum_w;
        m = -1;
      }
      if(err<minimal){
        minimal = err;
        cur_i = i;
        sel_m = m;
      }
    }
    if(minimal<cur_min){
      cur_min = minimal;
      cur_j = j;
      cur_theta = cur_i;
      cur_m = sel_m;
    }
  }
  pars_p->return_j = cur_j;
  pars_p->theta = usps[cur_j][cur_theta];
  pars_p->return_m = cur_m;
  return;
}

struct pars* AdaBoost(int B,float *alpha){
    struct pars* allPars = (struct pars*)malloc(sizeof(struct pars)*B);
    for (int b=0;b<B;b++){
        struct pars pars;
        train(&pars);
        // label = classify(X,pars)
        float *vec = usps[pars.return_j];
        float err = 0.0,w_sum = 0.0;
        for(int z =0;z<nums;z++){
            err += w[z] * ((vec[z]<=pars.theta) ^ (-pars.return_m == y[z]) );
            w_sum += w[z];
        }
        err = err/w_sum;
        alpha[b] = logf((1-err)/err);
        for(int z =0;z<nums;z++){
            w[z] = ((vec[z]<=pars.theta) ^ (-pars.return_m == y[z]))?(w[z] * (1-err) / err):w[z];
        }
        allPars[b].return_j = pars.return_j;
        allPars[b].return_m = pars.return_m;
        allPars[b].theta = pars.theta;
    }
    return allPars;
}

int * agg_class(float *alpha,struct pars* allPars,int B){
    float *res = (float *)malloc(sizeof(float)*nums);
    for (int z = 0; z < nums; ++z)
        res[z] = 0.0;
    int *c_hat = (int *)malloc(sizeof(int)*nums);
    for (int b = 0; b < B; ++b)
    {
        struct pars pars = allPars[b];
        float *vec = usps[pars.return_j];
        for(int z=0;z<nums;z++){
            res[z] += alpha[b]* ((vec[z]<=pars.theta)?(-pars.return_m):pars.return_m);
        }
    }
    for (int z = 0; z < nums; ++z)
        c_hat[z] = res[z]>= 0 ? 1:-1;
    free(res);
    return c_hat;
}

int main(){
    usps = (float **)malloc(sizeof(float *)*cols);
    w = (float*)malloc(sizeof(float)*nums);
    y = (int*)malloc(sizeof(int)*nums);;
    for (int i = 0; i < nums; ++i){
        w[i] = 1.0/nums;    
    }
    for(int j=0;j<cols;j++){
        usps[j] = (float *)malloc(sizeof(float)*nums);
    } 
    FILE* fp = fopen("uspsdata/uspsdata_ext.txt","r");
    FILE* fpcl = fopen("uspsdata/uspscl_ext.txt","r");
    for(int i=0;i<nums;i++){
        fscanf(fpcl,"%d",y+i);
        for(int j=0;j<cols;j++){
            fscanf(fp,"%f",*(usps+j)+i);
        }
    }
    fclose(fp);fclose(fpcl);

    /***********cuda here********/
    //cudaError_t err = cudaSuccess;
    hipError_t err = hipSuccess;

    // Print the vector length to be used, and compute its size
    int numElements = nums;
    size_t size = numElements * sizeof(float);
    printf("[Vector addition of %d elements]\n", numElements);

    // Allocate the host input vector A
    //float *h_A = (float *)malloc(size);
    float *h_A = usps[0];

    // Allocate the host input vector B
    //float *h_B = (float *)malloc(size);
    float *h_B = usps[1];

    // Allocate the host output vector C
    float *h_C = (float *)malloc(size);

    // Verify that allocations succeeded
    if (h_A == NULL || h_B == NULL || h_C == NULL)
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

    // Initialize the host input vectors
    for (int i = 0; i < numElements; ++i)
    {
        h_A[i] = rand()/(float)RAND_MAX;
        h_B[i] = rand()/(float)RAND_MAX;
    }

    // Allocate the device input vector A
    float *d_A = NULL;
    err = hipMalloc((void **)&d_A, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device input vector B
    float *d_B = NULL;
    err = hipMalloc((void **)&d_B, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device output vector C
    float *d_C = NULL;
    err = hipMalloc((void **)&d_C, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the host input vectors A and B in host memory to the device input vectors in
    // device memory
    printf("Copy input data from the host memory to the CUDA device\n");
    err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);
    err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the device result vector in device memory to the host result vector
    // in host memory.
    printf("Copy output data from the CUDA device to the host memory\n");
    err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    if (err != hipSuccess){
        fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    printf("float is %f\n", h_C[0]);
    err = hipFree(d_A);

    if (err != hipSuccess){
        fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_B);

    if (err != hipSuccess){
        fprintf(stderr, "Failed to free device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_C);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    // Reset the device and exit
    // cudaDeviceReset causes the driver to clean up all state. While
    // not mandatory in normal operation, it is good practice.  It is also
    // needed to ensure correct operation when the application is being
    // profiled. Calling cudaDeviceReset causes all profile data to be
    // flushed before the application exits
    err = hipDeviceReset();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    /*****************************/


    clock_t begin, end;
    double time_spent;
    begin = clock();
    struct pars* ap;
    float *alpha = (float *)malloc(sizeof(float)*5);;
    int *c_hat;
    ap = AdaBoost(5,alpha);
    c_hat = agg_class(alpha,ap,5);
    end = clock();
    time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
    for (int i = 0; i < 5; ++i)
    {
        printf("%d,%f,%d,%f\n",ap[i].return_j,ap[i].theta,ap[i].return_m,alpha[i]);    
    }
    printf("time is %f\n",time_spent);
    for(int j=0;j<cols;j++){
        free(usps[j]);
    }
    free(usps);
    free(w);
    free(y);
    free(alpha);
    free(ap);
    free(c_hat);
    return 0;
}
