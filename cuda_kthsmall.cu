#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <limits.h>

#define nums 200

void kthSmallest(int arr[], int k){
 	for (int i = 0; i < nums; ++i){
 		int upper_sum = 0,down_sum = 0,pivot = arr[i];
 		for (int j = 0; j < nums; ++j){
 			upper_sum += (pivot>arr[i]);
 			down_sum += (pivot>=arr[i]);
 		}
 		if (k<=down_sum && k>upper_sum){
	    	//printf("Found, %d\n",pivot);
	    }	
 	}
 	return;
}

void cuda_checker(hipError_t err,int i){
    if (err != hipSuccess){
        fprintf(stderr, "Failed to allocate device(error code %s and %d)!\n", hipGetErrorString(err),i);
        exit(EXIT_FAILURE);
    }
}
__global__ void
cuda_kthsmall(const int *x,const int k){
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < nums)
    {	
    	int upper_sum = 0,down_sum = 0,pivot = x[idx];
	    for (int i = 0; i < nums; ++i)
	    {
	    	upper_sum += (pivot>x[i]);
	    	down_sum += (pivot>=x[i]);
	    }
	    if (k<=down_sum && k>upper_sum)
	    {
	    	//printf("Found, %d\n",pivot);
	    }
    }
}


int main(){
    int *x = (int*)malloc(sizeof(int)*nums);
    time_t t;
    clock_t begin,end;
    double time_spend;
    int err_num = 0;
    srand((unsigned) time(&t));
    
    for (int i = 0; i < nums; ++i)
    {
        x[i] = rand() % 1000;
        //printf("%d ",x[i]);
    }
    printf("\n");
  
    begin = clock();
    kthSmallest(x,3);
    end = clock();
    time_spend = (double)(end - begin) / CLOCKS_PER_SEC;
    printf("CPU cost %f\n", time_spend);

    int *d_x = NULL;
    begin = clock();
    cuda_checker(hipMalloc((void **)&d_x,sizeof(int)*nums),err_num++);
    cuda_checker(hipMemcpy(d_x, x, sizeof(int)*nums, hipMemcpyHostToDevice),err_num++);
    cuda_kthsmall<<<(nums + 256 - 1) / 256, 256>>>(d_x,3);
    cuda_checker(hipFree(d_x),err_num++);
    end = clock();
    time_spend = (double)(end - begin) / CLOCKS_PER_SEC;
    printf("GPU cost %f\n", time_spend);

    free(x);
    cuda_checker(hipDeviceReset(),err_num++);
    return 0;
}