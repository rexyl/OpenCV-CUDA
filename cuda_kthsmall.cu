#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <limits.h>

#define nums 200000

void swap(int *a, int *b){
    int temp = *a;
    *a = *b;
    *b = temp;
}
int partition(int arr[], int l, int r){
    int x = arr[r], i = l;
    for (int j = l; j <= r - 1; j++)
    {
        if (arr[j] <= x)
        {
            swap(&arr[i], &arr[j]);
            i++;
        }
    }
    swap(&arr[i], &arr[r]);
    return i;
}
int kthSmallest(int arr[], int l, int r, int k){
    // If k is smaller than number of elements in array
    if (k > 0 && k <= r - l + 1)
    {
        // Partition the array around last element and get
        // position of pivot element in sorted array
        int pos = partition(arr, l, r);
        
        // If position is same as k
        if (pos-l == k-1)
            return arr[pos];
        if (pos-l > k-1)  // If position is more, recur for left subarray
            return kthSmallest(arr, l, pos-1, k);
        
        // Else recur for right subarray
        return kthSmallest(arr, pos+1, r, k-pos+l-1);
    }
    
    // If k is more than number of elements in array
    return INT_MAX;
}
void cuda_checker(hipError_t err,int i){
    if (err != hipSuccess){
        fprintf(stderr, "Failed to allocate device(error code %s and %d)!\n", hipGetErrorString(err),i);
        exit(EXIT_FAILURE);
    }
}
__global__ void
cuda_kthsmall(const int *x,const int k){
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < nums)
    {	
    	int upper_sum = 0,down_sum = 0,pivot = x[idx];
	    for (int i = 0; i < nums; ++i)
	    {
	    	upper_sum += (pivot>x[i]);
	    	down_sum += (pivot>=x[i]);
	    }
	    if (k<=down_sum && k>upper_sum)
	    {
	    	printf("Found, %d\n",pivot);
	    }
    }
}


int main(){
    int *x = (int*)malloc(sizeof(int)*nums);
    time_t t;
    clock_t begin,end;
    double time_spend;
    int err_num = 0;
    srand((unsigned) time(&t));
    
    for (int i = 0; i < nums; ++i)
    {
        x[i] = rand() % 1000;
        //printf("%d ",x[i]);
    }
    printf("\n");
  
    begin = clock();
    printf("%d\n", kthSmallest(x,0,nums-1,3));
    end = clock();
    time_spend = (double)(end - begin) / CLOCKS_PER_SEC;
    printf("CPU cost %f\n", time_spend);

    int *d_x = NULL;
    begin = clock();
    cuda_checker(hipMalloc((void **)&d_x,sizeof(int)*nums),err_num++);
    cuda_checker(hipMemcpy(d_x, x, sizeof(int)*nums, hipMemcpyHostToDevice),err_num++);
    cuda_kthsmall<<<(nums + 256 - 1) / 256, 256>>>(d_x,3);
    cuda_checker(hipFree(d_x),err_num++);
    end = clock();
    time_spend = (double)(end - begin) / CLOCKS_PER_SEC;
    printf("GPU cost %f\n", time_spend);

    free(x);
    cuda_checker(hipDeviceReset(),err_num++);
    return 0;
}